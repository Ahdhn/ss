#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

#include "helper.h"

#include <hip/hip_runtime_api.h>

__global__ void test6(float* ptr, float* out)
{
    
    __shared__ float4 shareM[32];
    float4*           ptr_temp = (float4*)ptr;
    for (int i = 0; i < 10; i++) {
        if (threadIdx.x < 32) {
            shareM[threadIdx.x] = ptr_temp[threadIdx.x];
        }
        __syncwarp();
    }

    out[threadIdx.x] = ((float*)(shareM))[threadIdx.x];
}

int main(int argc, char** argv)
{
    size_t bytes = sizeof(float4) * 32;
    float *ptr, *out;
    CUDA_ERROR(hipMalloc((void**)&ptr, bytes));
    CUDA_ERROR(hipMalloc((void**)&out, bytes));

    CUDA_ERROR(hipMemset(ptr, 0, bytes));
    CUDA_ERROR(hipMemset(ptr, 1, bytes));

    CUDA_ERROR(hipProfilerStart());

    test6<<<1, 32>>>(ptr, out);

    CUDA_ERROR(hipDeviceSynchronize());

    CUDA_ERROR(hipProfilerStop());

    return 0;
}
