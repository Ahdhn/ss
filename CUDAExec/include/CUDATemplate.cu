#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gtest/gtest.h"

#include "helper.h"

#include <hip/hip_runtime_api.h>

#define nthreads 32

struct myFloat4
{
    float x, y, z, w;
};

__global__ void test6(float* ptr, float* out)
{

    __shared__ myFloat4 shareM[nthreads];
    myFloat4*           ptr_temp = (myFloat4*)ptr;
    // for (int i = 0; i < 10; i++) {
    if (threadIdx.x < nthreads) {
        ((float*)(shareM))[threadIdx.x] = ((float*)(ptr_temp))[threadIdx.x];
        //printf("\n Read T=%d => %p", threadIdx.x, &shareM[threadIdx.x]);
    }
    __syncwarp();
    //}

    out[threadIdx.x] = ((float*)(shareM))[threadIdx.x];
    //printf(
    //    "\n Write T=%d => %p", threadIdx.x, &((float*)(shareM))[threadIdx.x]);
}

int main(int argc, char** argv)
{

    size_t bytes = sizeof(myFloat4) * nthreads;
    float *ptr, *out;
    CUDA_ERROR(hipMalloc((void**)&ptr, bytes));
    CUDA_ERROR(hipMalloc((void**)&out, bytes));

    CUDA_ERROR(hipMemset(ptr, 0, bytes));
    CUDA_ERROR(hipMemset(ptr, 1, bytes));

    CUDA_ERROR(hipProfilerStart());

    test6<<<1, nthreads>>>(ptr, out);

    CUDA_ERROR(hipDeviceSynchronize());

    CUDA_ERROR(hipProfilerStop());

    return 0;
}
